#include <sstream>
#include <iostream>
#include <thrust/sort.h>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>

template <typename T>
void sort_in_gpu(T *vec, int N) {
    thrust::sort(vec, vec + N);
	// for (int i = 0; i < N; i++) {
	//     std::cout << vec[i] << std::endl;
	// }
}

template <typename T>
void map_array(pybind11::array_t<T> vec) {
    pybind11::buffer_info ha = vec.request();
	if (ha.ndim != 1) {
		std::stringstream strstr;
		strstr << "ha.ndim != 1" << std::endl;
		strstr << "ha.ndim: " << ha.ndim << std::endl;
		throw std::runtime_error(strstr.str());
	}
	std::cout << "Test-----------------" << std::endl;

	int size = ha.shape[0];
	
	T* ptr = reinterpret_cast<T*>(ha.ptr);
	sort_in_gpu(ptr, size);
}

/*
int main(int argc, char *argv[]) {

	int a = 100;
	std::cout << "a is: " << a << std::endl;

	double A[1000] = {9, 2, 3, 6, 4, 8, 9};
	sort_in_gpu(A, 1000);
	// std::cout << A << std::endl;
	// std::cout << A + 1 << std::endl;

    return 0;
}
*/

PYBIND11_MODULE(gpu_library, m) {
    m.def("gpu_sort", map_array<double>);
}
