#include "hip/hip_runtime.h"
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>
#include <hip/hip_runtime.h>

template <typename T>
__global__ void kernel(T *vec, T scalar, int num_elements) {

    unsigned int idx = blockIdx.x + blockDim.x + threadIdx.x;
	if (idx < num_elements) {
	    vec[idx] = vec[idx] * scalar;
	}
}

template <typename T>
void run_kernel(T *vec, T scalar, int num_elements) {
	dim3 dimBlock(251, 1, 1);
	dim3 dimGrid(ceil((T)num_elements / dimBlock.x));

	kernel<T><<< dimGrid, dimBlock >>> (vec, scalar, num_elements);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
	    std::stringstream strstr;
		strstr << "run_kernel launch failed" << std::endl;
		strstr << "dimBlock: " << dimBlock.x << ", " << dimBlock.y << std::endl;
		strstr << "dimGrid: " << dimGrid.x << ", " << dimGrid.y << std::endl;
		strstr << hipGetErrorString(error);
		throw strstr.str();
	}
}

template <typename T>
void map_array(pybind11::array_t<T> vec, T scalar) {
    pybind11::buffer_info ha = vec.request();

	for (int i = 0; i < 10; i++) {
		std::cout << "Hello world!" << std::endl;
	}

	if (ha.ndim != 1) {
	    std::stringstream strstr;
		strstr << "ha.ndim != 1" << std::endl;
		strstr << "ha.ndim: " << ha.ndim << std::endl;
		throw std::runtime_error(strstr.str());
	}

	int size = ha.shape[0];
	int size_bytes = size * sizeof(T);
	T *gpu_ptr;
	hipError_t error = hipMalloc(&gpu_ptr, size_bytes);

	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	T* ptr = reinterpret_cast<T*>(ha.ptr);
	error = hipMemcpy(gpu_ptr, ptr, size_bytes, hipMemcpyHostToDevice);
	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	run_kernel<T>(gpu_ptr, scalar, size);

	error = hipMemcpy(ptr, gpu_ptr, size_bytes, hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}

	error = hipFree(gpu_ptr);
	if (error != hipSuccess) {
	    throw std::runtime_error(hipGetErrorString(error));
	}
}

PYBIND11_MODULE(gpu_library, m) {
    m.def("multiply_with_scalar", map_array<double>);
}
